#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <helper_functions.h>

//Rounded up
//for CUDA network setting
int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

// device fonction
//using HSV color map to find color in green
__device__ unsigned char binHSV(int R, int G, int B,float hL,float hH, float sL, float sH,float vL, float vH)
{
    //hL = 70,  hH = 160,  sL = 43, sH = 255,  vL = 46,  vH = 255
    // to find green color, G must be max (H in 120+-60)
    if (G > B&&G > R)
    {
        float V = G;      //V=max(R,G，B)  0-255
        if (V >vL && V <vH)  //for green V in 46-255
        {
            float cMin = min(R, B);
            float band = V - cMin;
            float S = band / V * 255; //s=(1-min(R,G,B)/max) 0-255
            if (S >sL && S <sH)  //for green S in 43-255
            {
                float H = (B - R) / band * 60 + 120;  //0-360
                if (H>hL && H<hH) //H in 120+-60
                    return 0; // black
            }
        }
    }
    return 255; //white
}

// kenerl fonction for CUDA network
__global__ void binKernel(unsigned char*indata, unsigned char*outdata, int w, int h,  float hL, float hH, float sL, float sH, float vL, float vH)
{
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;

    if (x < w&&y < h)
    {

        int coordinator = y * w * 3 + x * 3;
        int coordinator_out = y * w + x;

        //in opencv RGB chanels order is BGR
        //attention if BGR2RGB is used
        int r = indata[coordinator];
        int g = indata[coordinator + 1];
        int b = indata[coordinator + 2];

        outdata[coordinator_out] = binHSV(r, g, b,hL,hH,  sL, sH, vL, vH);
    }
}







extern "C" void binGreen(unsigned char*indata_h, unsigned char*outdata_h, int w, int h, float hL, float hH, float sL, float sH, float vL, float vH)
{

    int framesize = sizeof(unsigned  char)*w*h * 3;
    int framesizeout = sizeof(unsigned  char)*w*h;

    //device memory
    unsigned char*indata_d;
    checkCudaErrors(hipMalloc((void **)&indata_d, framesize));
    unsigned char*outdata_d;
    checkCudaErrors(hipMalloc((void **)&outdata_d, framesizeout));


    dim3 block(16, 16, 1); // 16x16 threads each block
    dim3 grid(iDivUp(w, block.x), iDivUp(h, block.y), 1); //nb block depends on image data

    //copy data from host to device
    checkCudaErrors(hipMemcpy(indata_d, indata_h, framesize, hipMemcpyHostToDevice));
    //call kernel function
    binKernel << <grid, block >> > (indata_d, outdata_d, w,h , hL, hH, sL, sH, vL, vH);

    //waiting for kernel function complete
    checkCudaErrors(hipDeviceSynchronize());

    // data from device to host
    checkCudaErrors(hipMemcpy(outdata_h, outdata_d, framesizeout, hipMemcpyDeviceToHost));
    //free memory
    hipFree(indata_d);
    hipFree(outdata_d);
    hipDeviceReset();
}


